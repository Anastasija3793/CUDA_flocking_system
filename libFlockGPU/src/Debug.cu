#include "Debug.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>

void cudaErrorPrint()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "Thrust allocation failed, error: " << hipGetErrorString(err) << "\n";
        
    }
}