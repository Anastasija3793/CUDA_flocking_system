#include "hip/hip_runtime.h"
#include "FlockGPU.cuh"
#include "BoidGPUKernels.cuh"
#include "Debug.cuh"

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <iostream>
#include <sstream>
#include <fstream>
#include <sys/time.h>
#include "random.cuh"


#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>

//for rand
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
return EXIT_FAILURE;}} while(0)


//3-tuple to store 3d vector type
typedef thrust::tuple<float,float,float> Float3;

struct get3dVec
{
    __host__ __device__ float3 operator()(Float3 a)
    {
        float x = thrust::get<0>(a);
        float y = thrust::get<1>(a);
        float z = thrust::get<2>(a);

        return make_float3(x,y,z);
    }
};

//struct random_float3
//{
//    __host__ __device__ float3 operator()(float3 v)
//    {
//        thrust::device_vector <float> tmp_PosPnts(NUM_BOIDS);
//        float * tmp_PosPnts_ptr = thrust::raw_pointer_cast(&tmp_PosPnts[0]);
//        randFloats(tmp_PosPnts_ptr, NUM_BOIDS);

//        float x = v.x;
//        float y = v.y;
//        float z = v.z;

//        float randX = x*tmp_PosPnts;
//        float randY = y*tmp_PosPnts;
//        float randZ = z*tmp_PosPnts;
//        return make_float3(randX,randY,randZ);
//    }
//};

// Return a host vector with random values in the range [0,1)
thrust::host_vector<float> random_vector(const size_t N,
                                         unsigned int seed = thrust::default_random_engine::default_seed)
{
    thrust::default_random_engine rng(seed);
    thrust::uniform_real_distribution<float> u01(-1.0f, 1.0f);//0,1
    thrust::host_vector<float> temp(N);
    for(size_t i = 0; i < N; i++) {
        temp[i] = u01(rng);
    }
    return temp;
}

FlockGPU::FlockGPU(int _numBoids)
{
    m_numBoids=_numBoids;

    m_dPos.resize(m_numBoids);
    m_dPosX.resize(m_numBoids);
    m_dPosY.resize(m_numBoids);
    m_dPosZ.resize(m_numBoids);

    m_dVel.resize(m_numBoids);
    m_dVelX.resize(m_numBoids);
    m_dVelY.resize(m_numBoids);
    m_dVelZ.resize(m_numBoids);

    m_pos.resize(m_numBoids);
    xTest.resize(m_numBoids);
    yTest.resize(m_numBoids);
    zTest.resize(m_numBoids);

    thrust::device_vector <float> myrand(NUM_BOIDS*3);
    //float * myrand_ptr = thrust::raw_pointer_cast(&myrand[0]);
    //randFloats(myrand_ptr, NUM_BOIDS*3);


//    myrand=random_vector(m_numBoids);
//    for(int i =0; i<m_numBoids; i++)
//    {
//        m_dPos=make_float3(myrand);
//    }


    m_dPosX=random_vector(m_numBoids);
    m_dPosY=random_vector(m_numBoids);
    m_dPosZ=random_vector(m_numBoids);

    m_dVelX=random_vector(m_numBoids);
    m_dVelY=random_vector(m_numBoids);
    m_dVelZ=random_vector(m_numBoids);




//    make_float3(tmp_PosPnts,tmp_PosPnts,tmp_PosPnts);

//    typedef thrust::device_vector<float>::iterator                     FloatIterator;
//    typedef thrust::tuple<FloatIterator, FloatIterator, FloatIterator> FloatIteratorTuple;
//    typedef thrust::zip_iterator<FloatIteratorTuple>                   Float3Iterator;

//    Float3Iterator pos_first = thrust::make_zip_iterator(make_tuple(m_dPosX.begin(), m_dPosY.begin(), m_dPosZ.begin()));
//    Float3Iterator pos_last  = thrust::make_zip_iterator(make_tuple(m_dPosX.end(), m_dPosY.end(), m_dPosZ.end()));
//    Float3Iterator vel_first = thrust::make_zip_iterator(make_tuple(m_dVelX.begin(), m_dVelY.begin(), m_dVelZ.begin()));
//    Float3Iterator vel_last  = thrust::make_zip_iterator(make_tuple(m_dVelX.end(), m_dVelY.end(), m_dVelZ.end()));

//    thrust::transform(pos_first, pos_last, m_dPos.begin(), get3dVec());
//    thrust::transform(vel_first, vel_last, m_dVel.begin(), get3dVec());


    thrust::transform(thrust::make_zip_iterator(make_tuple(m_dPosX.begin(), m_dPosY.begin(), m_dPosZ.begin())),
                      thrust::make_zip_iterator(make_tuple(m_dPosX.end(),   m_dPosY.end(),   m_dPosZ.end())),
                      m_dPos.begin(),
                      get3dVec());
    thrust::transform(thrust::make_zip_iterator(make_tuple(m_dVelX.begin(), m_dVelY.begin(), m_dVelZ.begin())),
                      thrust::make_zip_iterator(make_tuple(m_dVelX.end(),   m_dVelY.end(),   m_dVelZ.end())),
                      m_dVel.begin(),
                      get3dVec());

    m_dPosPtr = thrust::raw_pointer_cast(&m_dPos[0]);
    m_dVelPtr = thrust::raw_pointer_cast(&m_dVel[0]);

//    m_dPosPtr = thrust::raw_pointer_cast(m_dPos.data());
//    m_dVelPtr = thrust::raw_pointer_cast(m_dVel.data());
}

FlockGPU::~FlockGPU()
{

}

void FlockGPU::update()
{

    //N - blocks; M - threads
    unsigned int M = 1024;
    unsigned int N = m_numBoids/M + 1;

    //thrust::fill(m_dTarget.begin(), m_dTarget.begin()+m_numBoids,0);
    //thrust::fill(m_dPosPtr,m_dPosPtr + m_numBoids,0);

    //steerKernel<<<N,M>>>(m_dPosPtr,m_dVelPtr,m_dTargetPtr,m_dTargetPtr);
    //hipDeviceSynchronize();

    updateKernel<<<N,M>>>(m_dPosPtr,m_dVelPtr);
    hipDeviceSynchronize();

    thrust::copy(m_dPos.begin(),m_dPos.end(),m_pos.begin());

    //print
//    thrust::copy(m_dPosX.begin(),m_dPosX.end(),xTest.begin());
//    thrust::copy(m_dPosY.begin(),m_dPosY.end(),yTest.begin());
//    thrust::copy(m_dPosZ.begin(),m_dPosZ.end(),zTest.begin());
//    std::cout<<"x: "<<m_pos[0].x<<'\n';
//    std::cout<<"y: "<<m_pos[0].y<<'\n';
//    std::cout<<"z: "<<m_pos[0].z<<'\n';
//    std::cout<<"x: "<<xTest[0]<<'\n';
//    std::cout<<"y: "<<yTest[0]<<'\n';
//    std::cout<<"z: "<<zTest[0]<<'\n';
//    printf("%d \n", m_dPosX[0]);

}

// From: https://github.com/NCCA/cuda_workshops/blob/master/shared/src/random.cu
/**
 * Fill an array with random floats using the CURAND function.
 * \param devData The chunk of memory you want to fill with floats within the range (0,1]
 * \param n The size of the chunk of data
 */
int FlockGPU::randFloats(float *&devData, const size_t n)
{
    // The generator, used for random numbers
    hiprandGenerator_t gen;

    // Create pseudo-random number generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

    // Set seed to be the current time (note that calls close together will have same seed!)
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

    // Generate n floats on device
    CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

    // Cleanup
    CURAND_CALL(hiprandDestroyGenerator(gen));
    return EXIT_SUCCESS;
}

void FlockGPU::dumpGeo(uint _frameNumber)
{
    char fname[150];

    std::sprintf(fname,"geo/flock_gpu.%03d.geo",++_frameNumber);
    // we will use a stringstream as it may be more efficient
    std::stringstream ss;
    std::ofstream file;
    file.open(fname);
    if (!file.is_open())
    {
        std::cerr << "failed to Open file "<<fname<<'\n';
        exit(EXIT_FAILURE);
    }
    // write header see here http://www.sidefx.com/docs/houdini15.0/io/formats/geo
    ss << "PGEOMETRY V5\n";
    ss << "NPoints " << m_numBoids << " NPrims 1\n";
    ss << "NPointGroups 0 NPrimGroups 1\n";
    // this is hard coded but could be flexible we have 1 attrib which is Colour
    ss << "NPointAttrib 1  NVertexAttrib 0 NPrimAttrib 2 NAttrib 0\n";
    // now write out our point attrib this case Cd for diffuse colour
    ss <<"PointAttrib \n";
    // default the colour to white
    ss <<"Cd 3 float 1 1 1\n";
    // now we write out the particle data in the format
    // x y z 1 (attrib so in this case colour)
    for(unsigned int i=0; i<m_numBoids; ++i)
    {


        ss<<m_pos[i].x<<" "<<m_pos[i].y<<" "<<m_pos[i].z << " 1 ";
        //ss<<"("<<_boids[i].cellCol.x<<" "<<_boids[i].cellCol.y<<" "<< _boids[i].cellCol.z<<")\n";
        ss<<"("<<std::abs(1)<<" "<<std::abs(1)<<" "<<std::abs(1)<<")\n";
    }

    // now write out the index values
    ss<<"PrimitiveAttrib\n";
    ss<<"generator 1 index 1 location1\n";
    ss<<"dopobject 1 index 1 /obj/AutoDopNetwork:1\n";
    ss<<"Part "<<m_numBoids<<" ";
    for(size_t i=0; i<m_numBoids; ++i)
    {
        ss<<i<<" ";
    }
    ss<<" [0	0]\n";
    ss<<"box_object1 unordered\n";
    ss<<"1 1\n";
    ss<<"beginExtra\n";
    ss<<"endExtra\n";
    // dump string stream to disk;
    file<<ss.rdbuf();
    file.close();
}
